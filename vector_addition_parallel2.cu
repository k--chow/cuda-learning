#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // think in parallel. threadIdx is parallel, as is blockIdx
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        out[tid] = a[tid] + b[tid];
    }
    

}

int main(){
    float *a, *b, *out; 
    float *a_gp, *b_gp, *out_gp;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&a_gp, sizeof(float) * N);
    hipMalloc((void**)&b_gp, sizeof(float) * N);
    hipMalloc((void**)&out_gp, sizeof(float) * N);


    // Transfer data
    //cudaMemcpy(void *dst, void *src, size_t count, cudaMemcpyKind kind)
    hipMemcpy(a_gp, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(b_gp, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Kernel function
    int sm=1; //i have 40. But I should do N/
    int block_size = 25; //block size or grid_size
    int grid_size = ((N + block_size) / block_size);

    vector_add<<<grid_size,block_size>>>(out_gp, a_gp, b_gp, N); 

    // Transfer data back
    hipMemcpy(out, out_gp, sizeof(float)*N, hipMemcpyDeviceToHost);

    // Verify
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // free memory
    hipFree(a_gp);
    hipFree(b_gp);
    hipFree(out_gp);

    free(a);
    free(b);
    free(out);
}